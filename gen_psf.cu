// -*- c++ -*-
/*

Input text file with space-separated x and y offset positions.
Define some parameters in code on number of frequency channels. 
TODO: include time

 */

#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
using std::cout;
using std::cerr;
using std::endl;
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <unistd.h>
#include <netdb.h>
#include <sys/socket.h>
#include <sys/types.h>
#include <netinet/in.h>
#include <time.h>
#include <arpa/inet.h>
#include <sys/syscall.h>
#include <syslog.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define CVAC 2.998e8
#define NANT 2016
#define NBASEL NANT*(NANT-1)/2
#define NCHAN 1
#define FCH1_MHZ 1350.0
#define CHBW_MHZ 64.0
#define NTHREADS_GPU 32
#define NSIDE 1024
#define PIX_ASEC 0.5
#define N_INTENSITY 8

// kernel to calculate uv coordinates from antenna positions
// run with NBASEL*NCH/NTHREADS_GPU/N_INTENSITY blocks of NTHREADS_GPU threads
__global__
void calc_uv(int *a1, int *a2, float *ant_x, float *ant_y, float *freqs, float *u, float *v) {

  int idx = N_INTENSITY*(blockIdx.x*NTHREADS_GPU + threadIdx.x);
  int bidx,chidx;
  
  for (int i=idx;i<idx+N_INTENSITY;i++) {
  
    bidx = (int)(i / NCHAN);
    chidx = (int)(i % NCHAN);

    u[i] = (freqs[chidx]/CVAC)*(ant_x[a2[bidx]]-ant_x[a1[bidx]]);
    v[i] = (freqs[chidx]/CVAC)*(ant_y[a2[bidx]]-ant_y[a1[bidx]]);
  }
    
}

// kernel to calculate PSF
// needs to integrate over UVs to get image. 
// run with NSIDE*NSIDE/NTHREADS_GPU blocks of NTHREADS_GPU threads
__global__
void calc_psf(float *u, float *v, float *psf) {

  int idx = blockIdx.x*NTHREADS_GPU + threadIdx.x;
  int li = (int)(idx/NSIDE);
  int mi = (int)(idx % NSIDE);

  float l = (li*1.-NSIDE/2.)*((PIX_ASEC/3600.)*M_PI/180.);
  float m = (mi*1.-NSIDE/2.)*((PIX_ASEC/3600.)*M_PI/180.);

  for (int i=0;i<NBASEL*NCHAN;i++) {
    if (u[i]*u[i] + v[i]*v[i] > 900.) 
      psf[idx] += cosf(2.*M_PI*(u[i]*l+v[i]*m));
  }
  
}

// function to read in antenna positions
void read_ants(char * fnam, float *ant_x, float *ant_y) {

  FILE *fin;
  fin=fopen(fnam,"r");
  for (int i=0;i<NANT;i++)
    fscanf(fin,"%f %f\n",&ant_x[i],&ant_y[i]);
  fclose(fin);

}

// function to init freqs
void init_freqs(float * freqs) {

  for (int i=0;i<NCHAN;i++)
    freqs[i] = FCH1_MHZ*1e6+CHBW_MHZ*1e6*i;

}

// function to fill a1 and a2 arrays
void fill_aas(int *a1, int *a2) {

  int ii = 0;
  for (int i=0;i<NANT-1;i++) {
    for (int j=i+1;j<NANT;j++) {
      a1[ii] = i;
      a2[ii] = j;
      ii++;
    }
  }
}

void usage() {

  printf("gen_psf -f <antenna file name>\n");
  
}

int main(int argc, char **argv) {

  // command line arguments
  char * fnam = (char *)malloc(sizeof(char)*200);
  for (int i=1;i<argc;i++) {
    if (strcmp(argv[i],"-f")==0) {
      strcpy(fnam,argv[i+1]);
    }
    if (strcmp(argv[i],"-h")==0) {
      usage();
      exit(1);
    }
  }

  printf("Read all command line args\n");
  printf("NBASEL is %d\n",NBASEL);

  // define all host arrays
  float * ant_x = (float *)malloc(sizeof(float)*NANT);
  float * ant_y = (float *)malloc(sizeof(float)*NANT);
  int * a1  = (int *)malloc(sizeof(int)*NBASEL);
  int * a2  = (int *)malloc(sizeof(int)*NBASEL);
  float * freqs = (float *)malloc(sizeof(float)*NCHAN);
  float * psf = (float *)malloc(sizeof(float)*NSIDE*NSIDE);
  float * h_u = (float *)malloc(sizeof(float)*NCHAN*NBASEL);
  float * h_v = (float *)malloc(sizeof(float)*NCHAN*NBASEL);
  
  // define all device arrays
  float *d_ant_x, *d_ant_y, *d_freqs, *u, *v, *d_psf;
  int *d_a1, *d_a2;
  hipMalloc((void **)&d_ant_x, NANT*sizeof(float));
  hipMalloc((void **)&d_ant_y, NANT*sizeof(float));
  hipMalloc((void **)&d_freqs, NCHAN*sizeof(float));
  hipMalloc((void **)&u, NBASEL*NCHAN*sizeof(float));
  hipMalloc((void **)&v, NBASEL*NCHAN*sizeof(float));
  hipMalloc((void **)&d_psf, NSIDE*NSIDE*sizeof(float));
  hipMalloc((void **)&d_a1, NBASEL*sizeof(int));
  hipMalloc((void **)&d_a2, NBASEL*sizeof(int));

  // init ant pos
  read_ants(fnam,ant_x,ant_y);
  printf("Read ant pos\n");

  // init freqs
  init_freqs(freqs);
  printf("init_freqs\n");

  // init a1/a2
  fill_aas(a1,a2);
  printf("Fill a1 a2\n");

  // copy to device
  hipMemcpy(d_ant_x, ant_x, NANT*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_ant_y, ant_y, NANT*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_freqs, freqs, NCHAN*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_a1, a1, NBASEL*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_a2, a2, NBASEL*sizeof(int), hipMemcpyHostToDevice);
  printf("Copied to device\n");

  // run kernels
  calc_uv<<<NBASEL*NCHAN/NTHREADS_GPU/N_INTENSITY,NTHREADS_GPU>>>(d_a1, d_a2, d_ant_x, d_ant_y, d_freqs, u, v);
  hipDeviceSynchronize();
  printf("Calc uv\n");
  calc_psf<<<NSIDE*NSIDE/NTHREADS_GPU,NTHREADS_GPU>>>(u,v,d_psf);
  hipDeviceSynchronize();
  printf("Calc PSF\n");

  // copy to host
  hipMemcpy(psf, d_psf, NSIDE*NSIDE*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_u, u, NBASEL*NCHAN*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_v, v, NBASEL*NCHAN*sizeof(float), hipMemcpyDeviceToHost);
  printf("copied to host\n");
  
  // write to file
  FILE *fout;
  fout=fopen("tmp.dat","w");
  for (int i=0;i<NSIDE*NSIDE;i++) fprintf(fout,"%f\n",psf[i]);
  fclose(fout);
  fout=fopen("tmp_uv.dat","w");
  for (int i=0;i<NCHAN*NBASEL;i++) fprintf(fout,"%f %f\n",h_u[i],h_v[i]);
  fclose(fout);

  printf("writte to tmp.dat\n");
  
  free(ant_x);
  free(ant_y);
  free(a1);
  free(a2);
  free(psf);
  free(freqs);
  free(h_u);
  free(h_v);
  hipFree(d_ant_x);
  hipFree(d_ant_y);
  hipFree(d_freqs);
  hipFree(u);
  hipFree(v);
  hipFree(d_psf);
  hipFree(d_a1);
  hipFree(d_a2);
  
  free(fnam);
  

}
